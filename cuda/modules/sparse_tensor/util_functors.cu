#include "hip/hip_runtime.h"
#ifndef UTIL_FUNCTORS
#define UTIL_FUNCTORS

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>

#include <thrust/fill.h>
#include <thrust/device_vector.h>

template <typename T>
struct is_grade {
	is_grade(IndexType N, T grade) : grade_(grade), N_(N) {}
	T grade_;
	IndexType N_;

	__host__ __device__ bool operator()(const T &i) {
		if (grade_ == 0) {
			return i == 0;
		} else if (grade_ == 1) {
			return i > 0 && i <= N_;
		} else if (grade_ == 2) {
			return i > N_;
		}
		return false;
	}
};

template <typename T>
struct is_component {
	is_component(T component) : component_(component) {}
	T component_;

	__host__ __device__ bool operator()(const T &i) {
		return component_ == i;
	}
};

template <typename T>
struct compare_given_threshold {

	compare_given_threshold(CoeffType threshold) : threshold(threshold) {}
	CoeffType threshold;

  __host__ __device__ bool operator()(T x, T y) const {
	return abs(x-y) < threshold;
  }
};
#endif
