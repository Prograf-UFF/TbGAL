#include "hip/hip_runtime.h"
#ifndef UTILS_CU
#define UTILS_CU

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include "../metric/metric.cu"
#include "util_functors.cu"
#include <unistd.h>


enum Axis {
	I,
	J,
	K,
	EXISTS,
	VALUES
};

enum Operation {
	OUTER_PRODUCT,
	LEFT_CONTRACTION,
	RIGHT_CONTRACTION,
	DOT_PRODUCT
};


template <typename Iterator>
class repeated_range {
    public:

	    typedef typename thrust::iterator_difference<Iterator>::type difference_type;

	    struct repeat_functor : public thrust::unary_function<difference_type,difference_type> {
	        difference_type repeats;

	        repeat_functor(difference_type repeats) : repeats(repeats) {}

	        __host__ __device__ difference_type operator()(const difference_type& i) const {
	            return i / repeats;
	        }
	    };

	    typedef typename thrust::counting_iterator<difference_type> CountingIterator;
	    typedef typename thrust::transform_iterator<repeat_functor, CountingIterator> TransformIterator;
	    typedef typename thrust::permutation_iterator<Iterator,TransformIterator> PermutationIterator;

	    typedef PermutationIterator iterator;

	    repeated_range(Iterator first, Iterator last, difference_type repeats) : first(first), last(last), repeats(repeats) {}

	    iterator begin(void) const {
	        return PermutationIterator(first, TransformIterator(CountingIterator(0), repeat_functor(repeats)));
	    }

	    iterator end(void) const {
	        return begin() + repeats * (last - first);
	    }

    protected:
	    Iterator first;
	    Iterator last;
	    difference_type repeats;
};


template <typename Iterator>
class tiled_range {
    public:

	    typedef typename thrust::iterator_difference<Iterator>::type difference_type;

	    struct tile_functor : public thrust::unary_function<difference_type,difference_type> {
	        difference_type tile_size;

	        tile_functor(difference_type tile_size) : tile_size(tile_size) {}

	        __host__ __device__ difference_type operator()(const difference_type& i) const {
	            return i % tile_size;
	        }
	    };

	    typedef typename thrust::counting_iterator<difference_type> CountingIterator;
	    typedef typename thrust::transform_iterator<tile_functor, CountingIterator> TransformIterator;
	    typedef typename thrust::permutation_iterator<Iterator,TransformIterator> PermutationIterator;

	    typedef PermutationIterator iterator;

	    tiled_range(Iterator first, Iterator last, difference_type tiles) : first(first), last(last), tiles(tiles) {}

	    iterator begin(void) const {
	        return PermutationIterator(first, TransformIterator(CountingIterator(0), tile_functor(last - first)));
	    }

	    iterator end(void) const {
	        return begin() + tiles * (last - first);
	    }

    protected:
	    Iterator first;
	    Iterator last;
	    difference_type tiles;
};


template <typename T>
__host__ __device__ T getY(T idx, T N_) {
	return (T)(N_ - 2 - static_cast<T>(sqrt(-8 * idx + 4 * N_ *(N_ - 1) - 7) / 2.0 - 0.5));
}


template <typename T>
__host__ __device__ T getX(T idx, T N_) {
	auto Y = getY(idx, N_);
	return idx + Y + 1 - N_ * (N_ - 1) / 2 + (N_ - Y) * ((N_ - Y - 1)) / 2;
}


template <typename T>
__host__ __device__ T ji2idx(T j, T i, T N) {
	return ((N * (N - 1)) >> 1) - (((N - j) * (N - j - 1)) >> 1) + i;
}


template <typename T>
struct idx2ji {
	__host__ __device__ idx2ji(IndexType N) : N_(N + 1) {}
	T N_;
	__host__ __device__	thrust::tuple<T, T> operator()(const T& idx) const {
		if (idx == 0) {
			return thrust::make_tuple<T, T>(0, 0);
		}
		T idx_ = idx - 1;
		thrust::tuple<T, T> to_return = thrust::make_tuple<T, T>(getY(idx_, N_), getX(idx_, N_));
		return to_return;
	}
};


__host__ __device__ CoeffType canonical_sort(const IndexType &u, const IndexType &v, const IndexType &x, const IndexType &y) {
	IndexType sorter[4];
	sorter[0] = u;
	sorter[1] = v;
	sorter[2] = x;
	sorter[3] = y;

	int count_changes = 0;
	bool changed = false;
	do {
		changed = false;
		for (int i = 0; i < 3; i++) {
			if ((sorter[i] > sorter[i + 1])) {
				IndexType aux = sorter[i];
				sorter[i] = sorter[i + 1];
				sorter[i + 1] = aux;
				changed = true;
				if (sorter[i] != 0 && sorter[i + 1] != 0) {
					count_changes++;
				}
			}
		}
	} while (changed);
	return static_cast<CoeffType>(pow(-1, count_changes));
};


__host__ __device__ IndexType output_basis(const IndexType &i, const IndexType &j, const IndexType &N) {
	if (i == 0) {
		return ji2idx<IndexType>(i, j, N);
	} else if (j == 0) {
		return ji2idx<IndexType>(j, i, N);
	} else if (i < j) {
		return ji2idx<IndexType>(i, j, N);
	} else if (i > j){
		return ji2idx<IndexType>(j, i, N);
	} else {
		return static_cast<IndexType>(0);
	}
};

template < class MetricType, typename ReturnType, class = typename std::enable_if<std::is_base_of<Metric, MetricType>::value>::type>
struct GeometricProductTensorFunctor {
	GeometricProductTensorFunctor(IndexType N, const MetricType &metric, const Axis &axis) : metric(metric), axis(axis), N_(N) {}

	short axis;
	MetricType metric;
	IndexType N_;

	__host__ __device__ ReturnType operator() (const thrust::tuple<IndexType, IndexType> &it) {
		IndexType j = thrust::get<0>(it);
		IndexType i = thrust::get<1>(it);

		if (axis == Axis::I) {
			return i;
		} else if (axis == Axis::J) {
			return j;
		}

		IndexType K = 0;
		CoeffType val = 0;

		auto my_functor = idx2ji<IndexType>(N_);
		thrust::tuple<IndexType, IndexType> basis_i = my_functor(i);
		thrust::tuple<IndexType, IndexType> basis_j = my_functor(j);

		IndexType u = thrust::get<0>(basis_j);
		IndexType v = thrust::get<1>(basis_j);
		IndexType x = thrust::get<0>(basis_i);
		IndexType y = thrust::get<1>(basis_i);

		if (i == 0) {
			K = j;
			val = 1;
		} else if (j == 0) {
			K = i;
			val = 1;
		} else {
			if (i == j) {
				K = 0;
				val = canonical_sort(u, v, x, y) * metric.metric_factor(u, v);
			} else if (u == x) {
				K = output_basis(v, y, N_);
				val = canonical_sort(u, v, x, y) * metric.diagonal_entry(u); // vy
			} else if (u == y) {
				K = output_basis(v, x, N_);
				val = canonical_sort(u, v, x, y) * metric.diagonal_entry(u); // vx
			} else if (v == x) {
				K = output_basis(u, y, N_);
				val = canonical_sort(u, v, x, y) * metric.diagonal_entry(v); // uy
			} else if (v == y) {
				K = output_basis(u, x, N_);
				val = canonical_sort(u, v, x, y) * metric.diagonal_entry(v); // ux
			}
		}

		if (axis == Axis::K) {
			return K;
		} else if (axis == Axis::VALUES) {
			return val;
		} else if (axis == Axis::EXISTS) {
			return val != 0;
		}
		return 0;
	}
};

template < class MetricType, typename ReturnType, class = typename std::enable_if<std::is_base_of<Metric, MetricType>::value>::type>
struct NONGeometricProductTensorFunctor {
	NONGeometricProductTensorFunctor(IndexType N, const MetricType &metric, const Axis &axis) : metric(metric), axis(axis), N_(N) {}

	short axis;
	MetricType metric;
	IndexType N_;

	__host__ __device__ ReturnType operator() (const thrust::tuple<IndexType, IndexType> &it) {
		IndexType j = thrust::get<0>(it);
		IndexType i = thrust::get<1>(it);

		if (axis == Axis::I) {
			return i;
		} else if (axis == Axis::J) {
			return j;
		}

		IndexType K = 0;
		CoeffType val = 0;

		auto my_functor = idx2ji<IndexType>(N_);
		thrust::tuple<IndexType, IndexType> basis_i = my_functor(i);
		thrust::tuple<IndexType, IndexType> basis_j = my_functor(j);

		IndexType u = thrust::get<0>(basis_j);
		IndexType v = thrust::get<1>(basis_j);
		IndexType x = thrust::get<0>(basis_i);
		IndexType y = thrust::get<1>(basis_i);

		if (i == 0) {
			K = j;
			val = 1;
		} else if (j == 0) {
			K = i;
			val = 1;
		} else {
			if (i == j) {
				K = 0;
				val = canonical_sort(u, v, x, y) * metric.metric_factor(u, v);
			} else if (u == x) {
				K = output_basis(v, y, N_);
				val = canonical_sort(u, v, x, y) * metric.diagonal_entry(u); // vy
			} else if (u == y) {
				K = output_basis(v, x, N_);
				val = canonical_sort(u, v, x, y) * metric.diagonal_entry(u); // vx
			} else if (v == x) {
				K = output_basis(u, y, N_);
				val = canonical_sort(u, v, x, y) * metric.diagonal_entry(v); // uy
			} else if (v == y) {
				K = output_basis(u, x, N_);
				val = canonical_sort(u, v, x, y) * metric.diagonal_entry(v); // ux
			}
		}

		if (axis == Axis::K) {
			return K;
		} else if (axis == Axis::VALUES) {
			return val;
		} else if (axis == Axis::EXISTS) {
			return val == 0;
		}
		return 0;
	}
};


// template < class MetricType, class = typename std::enable_if<std::is_base_of<Metric, MetricType>::value>::type>
// SparseTensor<IndexType, CoeffType, cusp::device_memory> build_geometric_product_tensor(IndexType N, MetricType metric) {
//
// 	IndexType full_size = (N * (N + 1) >> 1) + 1;
//
// 	cusp::array1d<IndexType, cusp::host_memory> all_basis = cusp::counting_array<IndexType>(full_size);
//
// 	typedef typename thrust::host_vector<IndexType>::iterator Iterator;
// 	repeated_range<Iterator> all_basis_repeated(all_basis.begin(), all_basis.end(), full_size);
// 	tiled_range<Iterator> all_basis_tiled(all_basis.begin(), all_basis.end(), full_size);
//
// 	typedef typename repeated_range<Iterator>::iterator repeated_it;
// 	typedef typename tiled_range<Iterator>::iterator tiled_it;
//
// 	typedef thrust::tuple<repeated_it, tiled_it> IteratorTuple;
// 	typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
//
//
// 	ZipIterator zbegin = thrust::make_zip_iterator(thrust::make_tuple(all_basis_repeated.begin(), all_basis_tiled.begin()));
// 	ZipIterator zend = thrust::make_zip_iterator(thrust::make_tuple(all_basis_repeated.begin(), all_basis_tiled.begin())) + (full_size * full_size);
//
// 	IndexType size = thrust::count_if<ZipIterator>(zbegin, zend, GeometricProductTensorFunctor<MetricType, bool>(N, metric, Axis::EXISTS));
//
// 	// std::cout << "FULL SIZE: " << full_size * full_size << std::endl;
// 	// std::cout << "SIZE: " << size << std::endl;
//
// 	cusp::array1d<thrust::tuple<IndexType, IndexType>, cusp::host_memory> tuples(size);
//
// 	thrust::copy_if(thrust::host, zbegin, zend, tuples.begin(), GeometricProductTensorFunctor<MetricType, bool>(N, metric, Axis::EXISTS));
//
// 	cusp::array1d<IndexType, cusp::host_memory> K(size);
// 	thrust::transform(thrust::host, tuples.begin(), tuples.end(),
// 		K.begin(),
// 		GeometricProductTensorFunctor<MetricType, IndexType>(N, metric, Axis::K));
//
// 	cusp::array1d<IndexType, cusp::host_memory> I(size);
// 	thrust::transform(thrust::host, tuples.begin(), tuples.end(),
// 		I.begin(),
// 		GeometricProductTensorFunctor<MetricType, IndexType>(N, metric, Axis::I));
//
// 	cusp::array1d<IndexType, cusp::host_memory> J(size);
// 	thrust::transform(thrust::host, tuples.begin(), tuples.end(),
// 		J.begin(),
// 		GeometricProductTensorFunctor<MetricType, IndexType>(N, metric, Axis::J));
//
//
// 	cusp::array1d<CoeffType, cusp::host_memory> values(size);
// 	thrust::transform(thrust::host, tuples.begin(), tuples.end(),
// 		values.begin(),
// 		GeometricProductTensorFunctor<MetricType, CoeffType>(N, metric, Axis::VALUES));
//
// 	// std::cout << "I: " << *(thrust::max_element(I.begin(), I.end())) << std::endl;
// 	// std::cout << "J: " << *(thrust::max_element(J.begin(), J.end())) << std::endl;
// 	// std::cout << "K: " << *(thrust::max_element(K.begin(), K.end())) << std::endl;
//
// 	std::vector<IndexType> shape = { full_size, full_size, full_size };
//
// 	FILE *pfile;
// 	pfile = fopen("/home/eduardovera/values.bin", "wb");
// 	fwrite(&(*values.begin()), sizeof(values), values.size(), pfile);
// 	fclose(pfile);
//
// 	cusp::array1d<IndexType, cusp::device_memory> I_dev(I.begin(), I.end());
// 	cusp::array1d<IndexType, cusp::device_memory> J_dev(J.begin(), J.end());
// 	cusp::array1d<IndexType, cusp::device_memory> K_dev(K.begin(), K.end());
// 	cusp::array1d<CoeffType, cusp::device_memory> values_dev(values.begin(), values.end());
//
//
// 	return SparseTensor<IndexType, CoeffType, cusp::device_memory>(I_dev, J_dev, K_dev, values_dev, shape);
// }


template < class MetricType, class = typename std::enable_if<std::is_base_of<Metric, MetricType>::value>::type>
SparseTensor<IndexType, CoeffType, cusp::device_memory> build_geometric_product_tensor(IndexType N, MetricType metric) {

	IndexType full_size = (N * (N + 1) >> 1) + 1;

	cusp::array1d<IndexType, cusp::device_memory> all_basis = cusp::counting_array<IndexType>(full_size);

	typedef typename thrust::device_vector<IndexType>::iterator Iterator;
	repeated_range<Iterator> all_basis_repeated(all_basis.begin(), all_basis.end(), full_size);
	tiled_range<Iterator> all_basis_tiled(all_basis.begin(), all_basis.end(), full_size);

	typedef typename repeated_range<Iterator>::iterator repeated_it;
	typedef typename tiled_range<Iterator>::iterator tiled_it;

	typedef thrust::tuple<repeated_it, tiled_it> IteratorTuple;
	typedef thrust::zip_iterator<IteratorTuple> ZipIterator;


	ZipIterator zbegin = thrust::make_zip_iterator(thrust::make_tuple(all_basis_repeated.begin(), all_basis_tiled.begin()));
	ZipIterator zend = thrust::make_zip_iterator(thrust::make_tuple(all_basis_repeated.begin(), all_basis_tiled.begin())) + (full_size * full_size);

	std::cout << "SIZE: " << thrust::distance(zbegin, zend) << std::endl;

	IndexType size = thrust::count_if<ZipIterator>(zbegin, zend, GeometricProductTensorFunctor<MetricType, bool>(N, metric, Axis::EXISTS));

	std::cout << "SIZE: " << size << std::endl;

	// cusp::array1d<thrust::tuple<IndexType, IndexType>, cusp::device_memory> tuples(size);
	// cusp::array1d<ZipIterator, cusp::device_memory> tuples(size);

	cusp::array1d<IndexType, cusp::device_memory> indices(size);

	thrust::counting_iterator<IndexType> first_index(0);

	thrust::counting_iterator<IndexType> last_index(thrust::distance(zbegin, zend));


	thrust::copy_if(first_index, last_index, zbegin, indices.begin(), GeometricProductTensorFunctor<MetricType, bool>(N, metric, Axis::EXISTS));
	// ZipIterator nzend = thrust::remove_copy_if(zbegin, zend, tuples.begin(), NONGeometricProductTensorFunctor<MetricType, bool>(N, metric, Axis::EXISTS));

	// std::cout << "SIZE: " << tuples.end() - tuples.begin() << std::endl;

	typedef thrust::device_vector<IndexType>::iterator IndexIterator;

	thrust::permutation_iterator<ZipIterator, IndexIterator> permiter_begin(zbegin, indices.begin());
	thrust::permutation_iterator<ZipIterator, IndexIterator> permiter_end(zbegin, indices.end());

	// delete indices?

	cusp::array1d<IndexType, cusp::device_memory> K(size);
	thrust::transform(permiter_begin, permiter_end,
		K.begin(),
		GeometricProductTensorFunctor<MetricType, IndexType>(N, metric, Axis::K));

	cusp::array1d<IndexType, cusp::device_memory> I(size);
	thrust::transform(permiter_begin, permiter_end,
		I.begin(),
		GeometricProductTensorFunctor<MetricType, IndexType>(N, metric, Axis::I));

	cusp::array1d<IndexType, cusp::device_memory> J(size);
	thrust::transform(permiter_begin, permiter_end,
		J.begin(),
		GeometricProductTensorFunctor<MetricType, IndexType>(N, metric, Axis::J));

	cusp::array1d<CoeffType, cusp::device_memory> values(size);
	thrust::transform(permiter_begin, permiter_end,
		values.begin(),
		GeometricProductTensorFunctor<MetricType, CoeffType>(N, metric, Axis::VALUES));

	// std::cout << "I: " << *(thrust::max_element(I.begin(), I.end())) << std::endl;
	// std::cout << "J: " << *(thrust::max_element(J.begin(), J.end())) << std::endl;
	// std::cout << "K: " << *(thrust::max_element(K.begin(), K.end())) << std::endl;

	std::vector<IndexType> shape = { full_size, full_size, full_size };

	return SparseTensor<IndexType, CoeffType, cusp::device_memory>(I, J, K, values, shape);
}



template <typename ReturnType>
struct get_all_grades_and_compare {
	IndexType N_;
	IndexType inv_N_FULL;
	IndexType N_FULL;
	Axis axis;
	Operation operation;
	get_all_grades_and_compare(IndexType N, Operation operation, Axis axis) {
		this->N_ = N;
		this->N_FULL = (N_ * (N_ - 1) >> 1);
		this->axis = axis;
		this->operation = operation;
	}

	__host__ __device__ ReturnType operator()(const thrust::tuple<IndexType, IndexType, IndexType, CoeffType> &t) {
		IndexType r = thrust::get<0>(t);
		IndexType c = thrust::get<1>(t);
		IndexType l = thrust::get<2>(t);

		int grade_j = (r == static_cast<IndexType>(0)) ? 0 : (r <= this->N_ ) ? 1 : 2;
		int grade_i = (c == static_cast<IndexType>(0)) ? 0 : (c <= this->N_)  ? 1 : 2;
		int grade_k = (l == static_cast<IndexType>(0)) ? 0 : (l <= this->N_)  ? 1 : 2;

		bool itFits = false;

		if ((operation == Operation::OUTER_PRODUCT) && (grade_i + grade_j == grade_k) ) {
			itFits = true;
		} else if ((operation == Operation::LEFT_CONTRACTION) && (grade_i - grade_j == grade_k) ) {
			itFits = true;
		} else if ((operation == Operation::RIGHT_CONTRACTION) && (grade_j - grade_i == grade_k) ) {
			itFits = true;
		} else if ((operation == Operation::DOT_PRODUCT) && (abs(grade_i - grade_j) == grade_k) ) {
			itFits = true;
		}

		if (itFits) {
			if (this->axis == Axis::J) {
				return r;
			} else if (this->axis == Axis::I) {
				return c;
			} else if (this->axis == Axis::K) {
				return l;
			} else if (this->axis == Axis::VALUES) {
				return thrust::get<3>(t);
			} else if (this->axis == Axis::EXISTS) {
				return true;
			}
		}
		return static_cast<ReturnType>(0);
	}
};

SparseTensor<IndexType, CoeffType, cusp::device_memory>* extract_tensor(IndexType N, SparseTensor<IndexType, CoeffType, cusp::device_memory> *tensor, const Operation &operation) {

	typedef typename cusp::array1d<IndexType, cusp::device_memory>::iterator Iterator_Index;
	typedef typename cusp::array1d<CoeffType, cusp::device_memory>::iterator Iterator_Coeff;

	typedef thrust::tuple<Iterator_Index, Iterator_Index, Iterator_Index, Iterator_Coeff> IteratorTuple;
	typedef thrust::zip_iterator<IteratorTuple> ZipIterator;

	ZipIterator zbegin = thrust::make_zip_iterator(thrust::make_tuple(tensor->J.begin(), tensor->I.begin(), tensor->K.begin(), tensor->data.begin()));
	ZipIterator zend = zbegin + tensor->data.size();

	IndexType size = thrust::count_if<ZipIterator>(zbegin, zend, get_all_grades_and_compare<bool>(N, operation, Axis::EXISTS));

	thrust::device_vector<thrust::tuple<IndexType, IndexType, IndexType, CoeffType>> tuples(size);
	thrust::copy_if(zbegin, zend, tuples.begin(), get_all_grades_and_compare<bool>(N, operation, Axis::EXISTS));

	cusp::array1d<CoeffType, cusp::device_memory> newdata(size);
	cusp::array1d<IndexType, cusp::device_memory> newc(size);
	cusp::array1d<IndexType, cusp::device_memory> newr(size);
	cusp::array1d<IndexType, cusp::device_memory> newl(size);

	thrust::transform(tuples.begin(), tuples.end(),
					&newr[0],
					get_all_grades_and_compare<IndexType>(N, operation, Axis::J));

	thrust::transform(tuples.begin(), tuples.end(),
					&newc[0],
					get_all_grades_and_compare<IndexType>(N, operation, Axis::I));

	thrust::transform(tuples.begin(), tuples.end(),
					&newl[0],
					get_all_grades_and_compare<IndexType>(N, operation, Axis::K));

	thrust::transform(tuples.begin(), tuples.end(),
					&newdata[0],
					get_all_grades_and_compare<CoeffType>(N, operation, Axis::VALUES));

	return new SparseTensor<IndexType, CoeffType, cusp::device_memory>(newc, newr, newl, newdata, tensor->getDenseShape());

}

#endif
