#include "hip/hip_runtime.h"
#ifndef SPARSE_TENSOR_CU
#define SPARSE_TENSOR_CU

#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/print.h>
#include <cusp/multiply.h>
#include <iostream>
#include <vector>
#include <cusp/functional.h>
#include <cusp/print.h>
#include <cusp/array1d.h>
#include <thrust/tuple.h>
#include <thrust/find.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <cusp/transpose.h>
#include <functional>
#include <numeric>
#include <cusp/elementwise.h>
#include <cusp/array2d.h>
#include "util_functors.cu"
#include <fstream>

template <typename T>
struct is_zero {
	__host__ __device__ bool operator()(const T &i) {
		return abs(i) <= 1e-4;
	}
};

//typedef long long unsigned int IndexType;

template<typename IndexType, typename DataType, class MemorySpace>
class SparseTensor {

private:
	typedef cusp::array1d<DataType, MemorySpace> dataArray_t;
	typedef cusp::array1d_view<typename dataArray_t::iterator> dataArray_view_t;
	typedef cusp::array1d<IndexType, MemorySpace> indexArray_t;
	typedef cusp::array1d_view<typename indexArray_t::iterator> indexArray_view_t;
	typedef cusp::coo_matrix<IndexType, DataType, MemorySpace> spmcore_t;


	short rank;
	bool initialized;
	std::vector<IndexType> dense_shape;

public:
	indexArray_t I;
	indexArray_t J;
	indexArray_t K;
	dataArray_t data;
	spmcore_t core;
	/* DEFAULT CONSTRUCTOR */
	SparseTensor() {
		this->initialized = false;
	}

	SparseTensor(const SparseTensor& s) {
		rank = s.getRank();
		core = s.getCore();
		dense_shape = s.getDenseShape();
		initialized = s.isInitialized();
	}
	// TODO change to copy_if
	template<typename T1, typename T2>
	T1 erase_zeros(T1 indices, T2 data, bool erase=true) {
		if (!erase) {
			return T1(indices.begin(), indices.end());
		}
		typename T1::iterator new_end_indices = thrust::remove_if(indices.begin(), indices.end(), &data[0], is_zero<DataType>());
		return T1(indices.begin(), new_end_indices);
	}

	/* RANK 1 CONSTRUCTOR */
	SparseTensor(indexArray_t &indices, dataArray_t &data, std::vector<IndexType> dense_shape, bool erase=true) {

		this->dense_shape = dense_shape;
		this->rank = 1;
		this->core.resize(dense_shape[0], dense_shape[1], dataArray_view_t(data).size());
		if (dense_shape[0] == 1) {
			this->core.column_indices = erase_zeros<indexArray_t, dataArray_t>(indices, data, erase);
			this->I = this->core.column_indices;
		}
		else if (dense_shape[1] == 1) {
			this->core.row_indices = erase_zeros<indexArray_t, dataArray_t>(indices, data, erase);
			this->J = this->core.row_indices;
		}
		else {
			throw std::invalid_argument("Wrong arguments for constructor");
		}
		this->core.values = erase_zeros<dataArray_t, dataArray_t>(data, data, erase);
		this->initialized = true;
	}

	/* RANK 2 CONSTRUCTOR */
	SparseTensor(indexArray_t &cols, indexArray_t &rows, dataArray_t &data, std::vector<IndexType> dense_shape) {

		// TODO assert values and N
		this->dense_shape = dense_shape;
		this->rank = 2;
		this->core.resize(dense_shape[0], dense_shape[1], data.size());
		this->core.column_indices = erase_zeros<indexArray_t, dataArray_t>(cols, data);
		this->core.row_indices = erase_zeros<indexArray_t, dataArray_t>(rows, data);

		this->I = this->core.column_indices;
		this->J = this->core.row_indices;

		this->core.values = erase_zeros<dataArray_t, dataArray_t>(data, data);
		this->initialized = true;
	}

	/* RANK 3 CONSTRUCTOR */
	SparseTensor(indexArray_t &cols, indexArray_t &rows, indexArray_t &layers, dataArray_t &data, std::vector<IndexType> dense_shape) {

		indexArray_t c = indexArray_t(erase_zeros<indexArray_t, dataArray_t>(cols, data));
		indexArray_t r = indexArray_t(erase_zeros<indexArray_t, dataArray_t>(rows, data));
		indexArray_t l = indexArray_t(erase_zeros<indexArray_t, dataArray_t>(layers, data));
		dataArray_t d = dataArray_t(erase_zeros<dataArray_t, dataArray_t>(data, data));

		this->I = c;
		this->J = r;
		this->K = l;
		this->data = d;

		// std::cout << "BEFORE: " << d.size() << std::endl;
//		std::ofstream myfile;
//		myfile.open("tensor.dat");
//		myfile << dense_shape[0] << std::endl;
//		for (IndexType i = 0; i < d.size(); i++) {
//			myfile << l[i] << ";" << r[i] << ";" << c[i] << ";" << d[i] << std::endl;
//		}
//		myfile.close();

		// TODO assert values and N
		this->dense_shape = dense_shape;
		this->rank = 3;
		this->core.resize(dense_shape[0], dense_shape[1] * dense_shape[2], d.size());
		this->core.row_indices = r;
		thrust::transform(l.begin(), l.end(), this->core.column_indices.begin(), cusp::multiplies_value<IndexType>(dense_shape[1]));
		thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(c.begin(), this->core.column_indices.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(c.begin(), this->core.column_indices.begin())) + this->core.column_indices.size(),
			this->core.column_indices.begin(),
			cusp::sum_pair_functor<IndexType>());
		this->core.values = d;
		// std::cout << "BEFORE: " << d.size() << std::endl;
		this->core.sort_by_row_and_column();
		this->initialized = true;

	}

	/* CORE CONSTRUCTOR */
	SparseTensor(const std::vector<IndexType>& dense_shape, short rank, const spmcore_t &core) {
		spmcore_t new_core(core);
		new_core.column_indices = erase_zeros<indexArray_t, dataArray_t>(core.column_indices, core.values);
		new_core.row_indices = erase_zeros<indexArray_t, dataArray_t>(core.row_indices, core.values);
		new_core.values = erase_zeros<dataArray_t, dataArray_t>(core.values, core.values);

		this->dense_shape = dense_shape;
		this->rank = rank;
		this->core = new_core;
		this->initialized = true;
	}

	/* DEFAULT DESTRUCTOR */
	~SparseTensor() {

	}

	template<typename IndexT, typename DataT, typename MemoryS>
	friend std::ostream& operator << (std::ostream &, const SparseTensor<IndexT, DataT, MemoryS> &);

	template<typename IndexT, typename DataT, typename MemoryS>
	friend SparseTensor<IndexT, DataT, MemoryS> operator* (const DataT &, const SparseTensor<IndexT, DataT, MemoryS> &);
//	template<typename IndexType, typename DataType, typename MemorySpace> friend SparseTensor<IndexType, DataType, MemorySpace> operator* (const SparseTensor<IndexType, DataType, MemorySpace> &, const DataType &);

	void print_shape() {
		std::cout << "( ";
		for (short i = 0; i < this->dense_shape.size(); i++) {
			std::cout << this->dense_shape[i] << " ";
		}
		std::cout << ")" << std::endl;
	}

	bool isInitialized() const {
		return this->initialized;
	}

	const std::vector<IndexType>& getDenseShape() const {
		return this->dense_shape;
	}

	const spmcore_t& getCore() const {
		return this->core;
	}

	const short& getRank() const {
		return this->rank;
	}

	DataType operator() (IndexType i) {
		if (this->getRank() == 1) {
			indexArray_t *array;
			if (this->dense_shape[0] == 1) {
				array = &core.column_indices;
			}
			else {
				array = &core.row_indices;
			}
			auto iter = thrust::find(thrust::device, array->begin(), array->end(), i);
			if (iter == array->end()) {
				return 0;
			}
			return core.values[iter - array->begin()];
		}
		else {
			throw std::logic_error("Not implemented for rank > 1 tensors");
		}
	}

	SparseTensor operator+ (const SparseTensor &lhs) const {
		spmcore_t new_core;

		cusp::add(this->getCore(), lhs.getCore(), new_core);
		return SparseTensor(this->getDenseShape(), this->getRank(), new_core);
	}
	SparseTensor operator- (const SparseTensor &lhs) const {

		spmcore_t new_core;
		cusp::subtract(this->getCore(), lhs.getCore(), new_core);
		return SparseTensor(this->getDenseShape(), this->getRank(), new_core);
	}

	bool operator==(const SparseTensor &t) {
		return (this->getCore().values.size() == t.getCore().values.size()
		&& thrust::equal(this->getCore().values.begin(), this->getCore().values.end(), t.getCore().values.begin(), compare_given_threshold<CoeffType>(epsilon))
		&& thrust::equal(this->getCore().column_indices.begin(), this->getCore().column_indices.end(), t.getCore().column_indices.begin())
		&& thrust::equal(this->getCore().row_indices.begin(), this->getCore().row_indices.end(), t.getCore().row_indices.begin()));
	}

	SparseTensor operator* (const SparseTensor &rhs) const {

		IndexType inner_dim_lhs = this->getDenseShape()[this->getDenseShape().size() - 1];
		IndexType inner_dim_rhs = rhs.getDenseShape()[0];

		if (inner_dim_lhs == inner_dim_rhs) {

			std::vector<IndexType> output_shape;
			output_shape.reserve(this->getDenseShape().size() + rhs.getDenseShape().size() - 2);
			output_shape.insert(output_shape.end(), this->getDenseShape().begin(), this->getDenseShape().begin() + this->getDenseShape().size() - 1);
			output_shape.insert(output_shape.end(), rhs.getDenseShape().begin() + 1, rhs.getDenseShape().end());

			short output_rank = output_shape.size();

			cusp::coo_matrix<IndexType, DataType, MemorySpace> output_core;

			cusp::coo_matrix<IndexType, DataType, MemorySpace> lhs_core = this->getCore();
//			cusp::sort_by_row(lhs_core.row_indices, lhs_core.column_indices, lhs_core.values);

			cusp::coo_matrix<IndexType, DataType, MemorySpace> rhs_core = rhs.getCore();
//			cusp::sort_by_row(rhs_core.row_indices, rhs_core.column_indices, rhs_core.values);
			// rhs_core.sort_by_row_and_column();
			// lhs_core.sort_by_row_and_column();

			cusp::multiply(lhs_core, rhs_core, output_core);


//			if (output_rank == 3) {
//				if (output_core.row_indices.size() > output_core.column_indices.size()) {
//					cusp::transpose(output_core, output_core);
//				}
//			}
			return SparseTensor(output_shape, output_rank - std::count(output_shape.begin(), output_shape.end(), 1), output_core);
		}
		else {
			throw std::invalid_argument("Incompatible shapes");
		}
	}

	/* Reshape */
	SparseTensor<IndexType, DataType, MemorySpace> reshape(std::vector<IndexType> new_shape) {
		IndexType current_n_elements = std::accumulate(std::begin(this->dense_shape), std::end(this->dense_shape), 1, std::multiplies<IndexType>());
		IndexType new_n_elements = std::accumulate(std::begin(new_shape), std::end(new_shape), 1, std::multiplies<IndexType>());

		spmcore_t core = this->getCore();

		IndexType w_old = this->getRank() == 3 ? this->getDenseShape()[1] + this->getDenseShape()[2] : this->getDenseShape()[1];
		IndexType w_new = new_shape.size() == 3 ? new_shape[1] + new_shape[2] : new_shape[1];

		cusp::array1d<IndexType, MemorySpace> idx(current_n_elements);
		cusp::array1d<IndexType, MemorySpace> row_indices(current_n_elements);
		cusp::array1d<IndexType, MemorySpace> column_indices(current_n_elements);
		cusp::array1d<DataType, MemorySpace> data = this->getCore().values;

		if (current_n_elements == new_n_elements) {

			thrust::transform(core.row_indices.begin(), core.row_indices.end(), idx.begin(), cusp::multiplies_value<IndexType>(w_old));
			thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(idx.begin(), core.column_indices.begin())),
				thrust::make_zip_iterator(thrust::make_tuple(idx.begin(), core.column_indices.begin())) + core.column_indices.size(),
				idx.begin(),
				cusp::sum_pair_functor<IndexType>());

			thrust::transform(idx.begin(), idx.end(), row_indices.begin(), cusp::divide_value<IndexType>(w_new));
			thrust::device_vector<IndexType> minus_jw(current_n_elements);
			thrust::transform(row_indices.begin(), row_indices.end(), minus_jw.begin(), cusp::multiplies_value<IndexType>(-w_new));
			thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(idx.begin(), minus_jw.begin())),
				thrust::make_zip_iterator(thrust::make_tuple(idx.begin(), minus_jw.begin())) + idx.size(),
				column_indices.begin(),
				cusp::sum_pair_functor<IndexType>());
		}
		else {
			throw std::invalid_argument("Incompatible shapes");
		}
		return SparseTensor<IndexType, DataType, MemorySpace>(column_indices, row_indices, data, new_shape);
	}

	/* Transpose */
	SparseTensor t() {
		if (this->getRank() < 3) {
			spmcore_t coreT;
			std::vector<IndexType> denseShapeT(this->getDenseShape().size());
			std::reverse_copy(this->dense_shape.begin(), this->dense_shape.end(), denseShapeT.begin());
			cusp::transpose(this->core, coreT);
			SparseTensor<IndexType, DataType, MemorySpace> tensorT(denseShapeT, this->getRank(), coreT);
			return tensorT;
		}
		throw std::logic_error("Not implemented");
	}

	static SparseTensor<IndexType, DataType, MemorySpace> tensor_dot(const SparseTensor &op1, const SparseTensor &op2, const SparseTensor &T) {


		IndexType inner_dim_op1 = op1.getDenseShape()[op1.getDenseShape().size() - 1];
		IndexType inner_dim_op1_T = T.getDenseShape()[0];
		IndexType inner_dim_T_op2 = T.getDenseShape()[T.getDenseShape().size() - 1];
		IndexType inner_dim_op2 = op2.getDenseShape()[0];

		// if (inner_dim_op1 == inner_dim_op1_T && inner_dim_T_op2 == inner_dim_op2) {
		// 	SparseTensor<IndexType, DataType, MemorySpace> Z = op1 * T;
		// 	std::vector<IndexType> new_shape = { op2.getDenseShape()[0], op2.getDenseShape()[0] };
		// 	SparseTensor<IndexType, DataType, MemorySpace> K = Z.reshape(new_shape);
		// 	auto a =  K * op2;
		// 	return a;
		// }


		if (inner_dim_op1 == inner_dim_op1_T && inner_dim_T_op2 == inner_dim_op2) {
			SparseTensor<IndexType, DataType, MemorySpace> Z = op1 * T;
			auto w = inner_dim_op1_T;
			cusp::array1d<IndexType, MemorySpace> c = Z.getCore().column_indices;
			cusp::array1d<IndexType, MemorySpace> I(c.size());
			cusp::array1d<IndexType, MemorySpace> J(c.size());
			cusp::array1d<DataType, MemorySpace> data(Z.getCore().values);

			thrust::transform(c.begin(), c.end(), I.begin(), cusp::modulus_value<IndexType>(w));
			thrust::transform(c.begin(), c.end(), J.begin(), cusp::divide_value<IndexType>(w));

			std::vector<IndexType> dims = {w, w};
			SparseTensor<IndexType, DataType, MemorySpace> Z_reshaped(I, J, data, dims);

			auto a =  Z_reshaped * op2;

			return a;
		}

		else {
			throw std::invalid_argument("Incompatible shapes");
		}
	}
};

template<typename IndexType, typename DataType, typename MemorySpace>
SparseTensor<IndexType, DataType, MemorySpace> operator* (const DataType &lhs, const SparseTensor<IndexType, DataType, MemorySpace> &rhs) {
	cusp::coo_matrix<IndexType, DataType, MemorySpace> new_core = rhs.getCore();
	thrust::transform(new_core.values.begin(), new_core.values.end(), new_core.values.begin(), cusp::multiplies_value<DataType>(lhs));
//	cusp::multiply(lhs, rhs.getCore(), new_core);
	return SparseTensor<IndexType, DataType, MemorySpace>(rhs.getDenseShape(), rhs.getRank(), new_core);
}

//template<typename IndexType, typename DataType, typename MemorySpace>
//SparseTensor<IndexType, DataType, MemorySpace> operator* (const SparseTensor<IndexType, DataType, MemorySpace> &lhs, const DataType &rhs) {
//	cusp::coo_matrix<IndexType, DataType, MemorySpace> new_core;
//	cusp::multiply(rhs, lhs.getCore(), new_core);
//	return SparseTensor<IndexType, DataType, MemorySpace>(lhs.getDenseShape(), lhs.getRank(), new_core);
//}


template<typename IndexType, typename DataType, typename MemorySpace>
std::ostream& operator << (std::ostream &os, const SparseTensor<IndexType, DataType, MemorySpace> &T) {

	if (T.getRank() == 3) {

		cusp::array1d<IndexType, MemorySpace> K(T.getCore().values.size());
		cusp::array1d<IndexType, MemorySpace> I(T.getCore().values.size());

		thrust::transform(T.getCore().column_indices.begin(), T.getCore().column_indices.end(), K.begin(), cusp::multiplies_value<IndexType>(1 / T.getDenseShape()[1]));

		cusp::array1d<IndexType, MemorySpace> kw(T.getCore().values.size());
		thrust::transform(K.begin(), K.end(), kw.begin(), cusp::multiplies_value<IndexType>(-T.getDenseShape()[1]));

		thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(T.getCore().column_indices.begin(), kw.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(T.getCore().column_indices.begin(), kw.begin())) + kw.size(),
			I.begin(),
			cusp::sum_pair_functor<IndexType>());

		os << "K" << '\t' << "J" << '\t' << "I" << '\t' << "VALUE" << std::endl;
		os << "-----------------------------" << std::endl;
		for (IndexType i = 0; i < K.size(); i++) {
			os << K[i] << '\t' << T.getCore().row_indices[i] << '\t' << I[i] << '\t' << T.getCore().values[i] << std::endl;
		}
	}
	return os;
}
#endif
