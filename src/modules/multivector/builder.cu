#ifndef BUILDER_CU
#define BUILDER_CU

#include "../../common.cu"
#include "../metric/metric.cu"
#include "multivector.cu"
#include "operations.cu"
#include <boost/python.hpp>
#include <vector>
#include <boost/python/object.hpp>
#include <boost/python/stl_iterator.hpp>


BOOST_PYTHON_MODULE(multivector) {
	namespace python = boost::python;
	namespace operations = MultivectorOperations;


	python::class_<Multivector, Multivector*>("Multivector", python::no_init)
		.def("__repr__", &Multivector::to_string)
		.def("set_N", &Multivector::set_N)
			.staticmethod("set_N")
		.def("get_N", &Multivector::get_N)
			.staticmethod("get_N")
		.def("get_N_FULL", &Multivector::get_N_FULL)
			.staticmethod("get_N_FULL")
		.def("getComponent", &Multivector::getComponent, python::return_value_policy<python::manage_new_object>())

		.def("get_component_max_projection", &Multivector::get_component_max_projection, python::return_value_policy<python::manage_new_object>())

		.def("__eq__", &operations::is_equals)

		.def("__pos__", &operations::UNARY_PLUS, python::return_value_policy<python::manage_new_object>())
		.def("__add__", &operations::ADD_SCALAR<int>, python::return_value_policy<python::manage_new_object>())
		.def("__radd__", &operations::ADD_SCALAR<int>, python::return_value_policy<python::manage_new_object>())
		.def("__add__", &operations::ADD_SCALAR<float>, python::return_value_policy<python::manage_new_object>())
		.def("__radd__", &operations::ADD_SCALAR<float>, python::return_value_policy<python::manage_new_object>())
		.def("__add__", &operations::ADD_SCALAR<double>, python::return_value_policy<python::manage_new_object>())
		.def("__radd__", &operations::ADD_SCALAR<double>, python::return_value_policy<python::manage_new_object>())
		.def("__add__", &operations::ADD, python::return_value_policy<python::manage_new_object>())
		.def("__radd__", &operations::ADD, python::return_value_policy<python::manage_new_object>())

		.def("__neg__", &operations::UNARY_MINUS, python::return_value_policy<python::manage_new_object>())

		.def("__sub__", &operations::SUB_SCALAR<int>, python::return_value_policy<python::manage_new_object>())
		.def("__rsub__", &operations::R_SUB_SCALAR<int>, python::return_value_policy<python::manage_new_object>())
		.def("__sub__", &operations::SUB_SCALAR<float>, python::return_value_policy<python::manage_new_object>())
		.def("__rsub__", &operations::R_SUB_SCALAR<float>, python::return_value_policy<python::manage_new_object>())
		.def("__sub__", &operations::SUB_SCALAR<double>, python::return_value_policy<python::manage_new_object>())
		.def("__rsub__", &operations::R_SUB_SCALAR<double>, python::return_value_policy<python::manage_new_object>())
		.def("__sub__", &operations::SUB, python::return_value_policy<python::manage_new_object>())
		.def("__rsub__", &operations::SUB, python::return_value_policy<python::manage_new_object>())

		.def("__mul__", &operations::PROD<int>, python::return_value_policy<python::manage_new_object>())
		.def("__rmul__", &operations::PROD<int>, python::return_value_policy<python::manage_new_object>())
		.def("__mul__", &operations::PROD<float>, python::return_value_policy<python::manage_new_object>())
		.def("__rmul__", &operations::PROD<float>, python::return_value_policy<python::manage_new_object>())
		.def("__mul__", &operations::PROD<double>, python::return_value_policy<python::manage_new_object>())
		.def("__rmul__", &operations::PROD<double>, python::return_value_policy<python::manage_new_object>())

		.def("__xor__", &operations::OP, python::return_value_policy<python::manage_new_object>())

		// .def("__invert__", &operations::REVERSE, python::return_value_policy<python::manage_new_object>())

		;


	python::def("e", &e, python::return_value_policy<python::manage_new_object>());
	python::def("generate_T", &generate_T<EuclideanMetric>);
	python::def("build_tensor", &build_tensor<EuclideanMetric>, python::return_value_policy<python::manage_new_object>());
	python::def("extract_tensor", &extract_tensor, python::return_value_policy<python::manage_new_object>());
	python::def("get_GP_T", &Multivector::get_GP_T, python::return_value_policy<python::manage_new_object>());

	python::def("REVERSE", &operations::REVERSE, python::return_value_policy<python::manage_new_object>());
	python::def("INVOLUTION", &operations::INVOLUTION, python::return_value_policy<python::manage_new_object>());
	python::def("CONJUGATE", &operations::CONJUGATE, python::return_value_policy<python::manage_new_object>());
	python::def("INVERSE", &operations::INVERSE, python::return_value_policy<python::manage_new_object>());

	python::def("GP", &operations::GP_tensor, python::return_value_policy<python::manage_new_object>());
	python::def("GP", &operations::GP, python::return_value_policy<python::manage_new_object>());

	python::def("LCONT", &operations::LCONT, python::return_value_policy<python::manage_new_object>());
	python::def("RCONT", &operations::RCONT, python::return_value_policy<python::manage_new_object>());

	python::def("DOT", &operations::dot, python::return_value_policy<python::manage_new_object>());

	python::def("SQR_NORM", &operations::SQR_NORM);//, python::return_value_policy<python::manage_new_object>());
	python::def("NORM", &operations::NORM);//, python::return_value_policy<python::manage_new_object>());
	python::def("IGP", &operations::IGP, python::return_value_policy<python::manage_new_object>());

	python::def("SCP", &operations::SCP_tensor, python::return_value_policy<python::manage_new_object>());
	python::def("SCP", &operations::SCP, python::return_value_policy<python::manage_new_object>());

	python::def("take_grade", &operations::take_grade, python::return_value_policy<python::manage_new_object>());
	python::def("fact_blade", &operations::FACT_BLADE);
	python::def("fact_versor", &operations::FACT_VERSOR);


}
#endif
