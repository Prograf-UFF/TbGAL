#ifndef BUILDER_CU
#define BUILDER_CU

#include "../../common.cu"
#include "../metric/metric.cu"
#include "multivector.cu"
#include "operations.cu"
#include <boost/python.hpp>
#include <vector>
#include <boost/python/object.hpp>
#include <boost/python/stl_iterator.hpp>


BOOST_PYTHON_MODULE(multivector) {
	namespace python = boost::python;
	namespace operations = MultivectorOperations;


	python::class_<Multivector>("Multivector", python::no_init)
		.def("__repr__", &Multivector::to_string)
		.def("set_N", &Multivector::set_N)
			.staticmethod("set_N")
		.def("get_N", &Multivector::get_N)
			.staticmethod("get_N")
		.def("get_N_FULL", &Multivector::get_N_FULL)
			.staticmethod("get_N_FULL")
		.def("getComponent", &Multivector::getComponent)
		.def("getGrade", &Multivector::get_grade_blade)
		.def("get_component_max_projection", &Multivector::get_component_max_projection)

		.def("__eq__", &operations::is_equals)

		.def("__pos__", &operations::UNARY_PLUS)
		.def("__add__", &operations::ADD_SCALAR<int>)
		.def("__radd__", &operations::ADD_SCALAR<int>)
		.def("__add__", &operations::ADD_SCALAR<float>)
		.def("__radd__", &operations::ADD_SCALAR<float>)
		.def("__add__", &operations::ADD_SCALAR<double>)
		.def("__radd__", &operations::ADD_SCALAR<double>)
		.def("__add__", &operations::ADD)
		.def("__radd__", &operations::ADD)

		.def("__neg__", &operations::UNARY_MINUS)

		.def("__sub__", &operations::SUB_SCALAR<int>)
		.def("__rsub__", &operations::R_SUB_SCALAR<int>)
		.def("__sub__", &operations::SUB_SCALAR<float>)
		.def("__rsub__", &operations::R_SUB_SCALAR<float>)
		.def("__sub__", &operations::SUB_SCALAR<double>)
		.def("__rsub__", &operations::R_SUB_SCALAR<double>)
		.def("__sub__", &operations::SUB)
		.def("__rsub__", &operations::SUB)

		.def("__mul__", &operations::PROD<int>)
		.def("__rmul__", &operations::PROD<int>)
		.def("__mul__", &operations::PROD<float>)
		.def("__rmul__", &operations::PROD<float>)
		.def("__mul__", &operations::PROD<double>)
		.def("__rmul__", &operations::PROD<double>)

		.def("__xor__", &operations::OP)

		// .def("__invert__", &operations::REVERSE, python::return_value_policy<python::manage_new_object>())

		;


	python::def("e", &e);
	python::def("generate_T", &generate_T<EuclideanMetric>);
	python::def("build_tensor", &build_tensor<EuclideanMetric>, python::return_value_policy<python::manage_new_object>());
	python::def("extract_tensor", &extract_tensor, python::return_value_policy<python::manage_new_object>());
	python::def("get_GP_T", &Multivector::get_GP_T, python::return_value_policy<python::manage_new_object>());

	python::def("REVERSE", &operations::REVERSE);
	python::def("INVOLUTION", &operations::INVOLUTION);
	python::def("CONJUGATE", &operations::CONJUGATE);
	python::def("INVERSE", &operations::INVERSE);

	python::def("GP", &operations::GP_tensor);
	python::def("GP", &operations::GP);

	python::def("LCONT", &operations::LCONT);
	python::def("RCONT", &operations::RCONT);

	python::def("DOT", &operations::dot);
	python::def("NATIVE", &operations::native);

	python::def("SQR_NORM", &operations::SQR_NORM);//, python::return_value_policy<python::manage_new_object>());
	python::def("NORM", &operations::NORM);//, python::return_value_policy<python::manage_new_object>());
	python::def("IGP", &operations::IGP);

	python::def("SCP", &operations::SCP_tensor);
	python::def("SCP", &operations::SCP);

	python::def("take_grade", &operations::take_grade);
	python::def("fact_blade", &operations::FACT_BLADE<python::list>);
	python::def("fact_versor", &operations::FACT_VERSOR<python::list>);
	python::def("set_device", &setDevice);

}
#endif
