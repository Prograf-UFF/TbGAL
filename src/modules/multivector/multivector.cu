#include "hip/hip_runtime.h"
#ifndef MULTIVECTOR_CU
#define MULTIVECTOR_CU

#include "../../common.cu"

#include "../sparse_tensor/SparseTensor.cu"
#include <cusp/print.h>
#include <cusp/array1d.h>
#include <thrust/tuple.h>
#include "../sparse_tensor/utils.cu"
#include <vector>
#include <chrono>
#include <type_traits>
#include "../metric/metric.cu"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <type_traits>
#include <set>

#include <boost/python.hpp>

SparseTensor<IndexType, CoeffType, MemorySpace> *GP_T;
SparseTensor<IndexType, CoeffType, MemorySpace> *OP_T;


class Multivector {
	private:
		SparseTensor<IndexType, CoeffType, MemorySpace> core;
		static IndexType N;
		static IndexType N_FULL;

	public:
		static IndexType get_N();
		static IndexType get_N_FULL();
		static void set_N(const IndexType N);
		static SparseTensor<IndexType, CoeffType, MemorySpace>* get_OP_T();
		static SparseTensor<IndexType, CoeffType, MemorySpace>* get_GP_T();

		Multivector();
		Multivector(const Multivector& m);
		Multivector(const SparseTensor<IndexType, CoeffType, MemorySpace> &core);
	    Multivector(IndexType index, CoeffType coeff = 1);

		SparseTensor<IndexType, CoeffType, MemorySpace> getCore() const;
		std::vector<int> get_grade() const;
		int get_grade_blade();
	    cusp::array1d<IndexType, MemorySpace> getComponentIndexes() const;

	    Multivector getComponent(IndexType idx);
		Multivector get_component_max_projection();

		// REVERSE
		Multivector operator ~();
		// UNARY PLUS
		Multivector operator +();
		// UNARY MINUS
		Multivector operator -();


		// SUM
		friend Multivector operator +(const Multivector &lhs, const Multivector &rhs);
		template<typename ScalarType, typename>
		friend Multivector operator +(const Multivector &lhs, const ScalarType &rhs);
		template<typename ScalarType, typename>
		friend Multivector operator +(const ScalarType &lhs, const Multivector &rhs);

		// DIFF
		friend Multivector operator -(const Multivector &lhs, const Multivector &rhs);
		template<typename ScalarType, typename>
		friend Multivector operator -(const Multivector &lhs, const ScalarType &rhs);
		template<typename ScalarType, typename>
		friend Multivector operator -(const ScalarType &lhs, const Multivector &rhs);

		// OUTER PRODUCT
		friend Multivector operator ^(const Multivector &lhs, const Multivector &rhs);

		// PRODUCT
		template<typename ScalarType, typename>
		friend Multivector operator *(const Multivector &lhs, const ScalarType &rhs);
		template<typename ScalarType, typename>
		friend Multivector operator *(const ScalarType &lhs, const Multivector &rhs);

		// OPERATOR ==
		friend bool operator ==(const Multivector &lhs, const Multivector &rhs);

		// OPERATOR <<
		friend std::ostream& operator <<(std::ostream& os, Multivector& m);


		// Multivector* REVERSE();
		// Multivector* INVOLUTION();
	    // Multivector* take_grade(IndexType grade);
	    std::string to_string();

};

IndexType Multivector::N = 0;
IndexType Multivector::N_FULL = 0;


/************************** END OF MULTIVECTOR.H ***************************/

/************************** GENERAL OPERATIONS.H ***************************/

template<typename MetricType>
SparseTensor<IndexType, CoeffType, MemorySpace> *build_tensor(MetricType metric);

template<typename MetricType>
void generate_T(const MetricType metric);

template<typename MetricType>
SparseTensor<IndexType, CoeffType, MemorySpace> *build_tensor(MetricType metric) {
  return new SparseTensor<IndexType, CoeffType, cusp::device_memory>(build_geometric_product_tensor<MetricType>(Multivector::get_N(), metric));
}

void setDevice(int device){
  hipSetDevice(device);
}

template<typename MetricType>
void generate_T(const MetricType metric) {
    GP_T = new SparseTensor<IndexType, CoeffType, MemorySpace>(build_geometric_product_tensor<MetricType>(Multivector::get_N(), metric));
}


/*************** END OF GENERAL OPERATIONS.H ****************/


/******************** MULTIVECTOR.CPP **********************/

IndexType Multivector::get_N() {
	return N;
}

IndexType Multivector::get_N_FULL() {
	return N_FULL;
}

void Multivector::set_N(const IndexType N) {
	Multivector::N = N;
	Multivector::N_FULL = (N * (N+1) >> 1) + 1;
}

Multivector::Multivector() {
	this->core = SparseTensor<IndexType, CoeffType, MemorySpace>();
}

Multivector::Multivector(const Multivector& m) {
	core = m.getCore();
}

Multivector::Multivector(const SparseTensor<IndexType, CoeffType, MemorySpace> &core) {
	this->core = core;
}

Multivector::Multivector(IndexType index, CoeffType coeff) {
	cusp::array1d<IndexType, MemorySpace> indices(1);
	cusp::array1d<CoeffType, MemorySpace> data(1);
	std::vector<IndexType> shape = { 1, Multivector::get_N_FULL()};

	data[0] = coeff;
	indices[0] = index;

	this->core = SparseTensor<IndexType, CoeffType, MemorySpace>(indices, data, shape, coeff == 0);

}

SparseTensor<IndexType, CoeffType, MemorySpace> Multivector::getCore() const {
	return this->core;
}

std::vector<int> Multivector::get_grade() const {
	std::vector<int> ret;
	if (core.getRank() == 1) {
		cusp::array1d<IndexType, MemorySpace> indices = core.getCore().column_indices;
		for (int i = 0; i <= 2; i++) {
			int count_grade = thrust::count_if(indices.begin(), indices.end(), is_grade<IndexType>(Multivector::get_N(), i));
			if (count_grade != 0) {
				ret.push_back(i);
			}
		}
	}
	// TODO else for handling exception
	return ret;
}

int Multivector::get_grade_blade() {
	return this->get_grade()[0];
}

cusp::array1d<IndexType, MemorySpace> Multivector::getComponentIndexes() const {
	return this->getCore().getCore().column_indices;
}

Multivector Multivector::getComponent(IndexType idx) {
	cusp::array1d<IndexType, MemorySpace> indices = this->getCore().getCore().column_indices;
	cusp::array1d<CoeffType, MemorySpace> values = this->getCore().getCore().values;

	cusp::array1d<IndexType, MemorySpace> new_indices(1);
	cusp::array1d<CoeffType, MemorySpace> new_values(1);

	thrust::copy_if(indices.begin(), indices.end(), &indices[0], &new_indices[0], is_component<IndexType>(idx));
	thrust::copy_if(values.begin(), values.end(), &indices[0], &new_values[0], is_component<IndexType>(idx));

	std::vector<IndexType> dense_shape = this->getCore().getDenseShape();

	SparseTensor<IndexType, CoeffType, MemorySpace> new_core(new_indices, new_values, dense_shape, false);
	return Multivector(new_core);
}

Multivector Multivector::get_component_max_projection() {
	cusp::array1d<IndexType, MemorySpace> indices = this->getComponentIndexes();
	cusp::array1d<CoeffType, MemorySpace> values = this->getCore().getCore().values;

	if (indices.size() == 0) {
		// throw
		// return NULL;
	}

	thrust::device_vector<CoeffType>::iterator iter = thrust::max_element(values.begin(), values.end());
	auto coeff_max = *iter;
	auto idx_max = iter - values.begin();

	iter = thrust::min_element(values.begin(), values.end());
	auto coeff_min = *iter;
	auto idx_min = iter - values.begin();

	if (abs(coeff_min) > coeff_max) {
		coeff_max = coeff_min;
		idx_max = idx_min;
	}

	// TODO fixop
	// return new Multivector(indices[idx_max]);
	return Multivector(indices[idx_max], coeff_max);
}

std::string Multivector::to_string() {
	std::string repr = "";
	bool first = true;
	if (core.getCore().values.size() == 0) {
		repr += "0";
	} else {
		idx2ji<IndexType> func(Multivector::get_N());
		for (IndexType i = 0; i < core.getCore().values.size(); i++) {
			if (core.getCore().column_indices[i] == 0) {
				repr += (core.getCore().values[i] > 0.0 ? (first ? "" : "+" ) : "") + std::to_string(core.getCore().values[i]);
				first = false;
			} else if (core.getCore().column_indices[i] <= Multivector::get_N()){
				repr += (core.getCore().values[i] > 0.0 ? (first ? "" : "+" ) : "") + (core.getCore().values[i] == 1.0 ? "" : std::to_string(core.getCore().values[i]) + "*") + "e(" + std::to_string(core.getCore().column_indices[i]) + ")";
				first = false;
			} else {
				thrust::tuple<IndexType, IndexType> basis = func(core.getCore().column_indices[i]);
				repr += (core.getCore().values[i] > 0.0 ? (first ? "" : "+" ) : "") + (core.getCore().values[i] == 1.0 ? "" : std::to_string(core.getCore().values[i]) + "*") + "e(" + std::to_string(thrust::get<0>(basis)) + ")^e(" + std::to_string(thrust::get<1>(basis)) + ")";
				first = false;
			}
		}
	}
	return repr;
}

Multivector e(IndexType index) {
  if (index > Multivector::get_N()) {
    throw std::logic_error("Can't allocate basis blade e(" + std::to_string(index) + ") in a " + std::to_string(Multivector::get_N()) + "-dimensional space");
  }
	return Multivector(index);
}

SparseTensor<IndexType, CoeffType, MemorySpace>* Multivector::get_GP_T() {
    return GP_T;
}

SparseTensor<IndexType, CoeffType, MemorySpace>* Multivector::get_OP_T() {;
    return OP_T;
}


#include "operations.cu"

// REVERSE
Multivector Multivector::operator ~() {
	return MultivectorOperations::REVERSE(*this);
}
// UNARY PLUS
Multivector Multivector::operator +() {
	return MultivectorOperations::UNARY_PLUS(*this);
}
// UNARY MINUS
Multivector Multivector::operator -() {
	return MultivectorOperations::UNARY_MINUS(*this);
}
// SUM
Multivector operator +(const Multivector &lhs, const Multivector &rhs) {
	return MultivectorOperations::ADD(lhs, rhs);
}
template<typename ScalarType>
Multivector operator +(const Multivector &lhs, const ScalarType &rhs) {
	return MultivectorOperations::ADD_SCALAR(lhs, rhs);
}
template<typename ScalarType>
Multivector operator +(const ScalarType &lhs, const Multivector &rhs) {
	return MultivectorOperations::ADD_SCALAR(rhs, lhs);
}
// DIFF
Multivector operator -(const Multivector &lhs, const Multivector &rhs) {
	return MultivectorOperations::SUB(lhs, rhs);
}
template<typename ScalarType>
Multivector operator -(const Multivector &lhs, const ScalarType &rhs) {
	return MultivectorOperations::SUB_SCALAR(lhs, rhs);
}
template<typename ScalarType>
Multivector operator -(const ScalarType &lhs, const Multivector &rhs) {
	return MultivectorOperations::SUB_SCALAR(rhs, lhs);
}
// OUTER PRODUCT
Multivector operator ^(const Multivector &lhs, const Multivector &rhs) {
	return MultivectorOperations::OP(lhs, rhs);
}
// PRODUCT
template<typename ScalarType>
Multivector operator *(const Multivector &lhs, const ScalarType &rhs) {
	return MultivectorOperations::PROD(lhs, rhs);
}
template<typename ScalarType>
Multivector operator *(const ScalarType &lhs, const Multivector &rhs) {
	return MultivectorOperations::PROD(rhs, lhs);
}
// OPERATOR ==
bool operator ==(const Multivector &lhs, const Multivector &rhs) {
	return MultivectorOperations::is_equals(lhs, rhs);
}
// OPERATOR <<
std::ostream& operator <<(std::ostream& os, Multivector& m) {
	os << m.to_string();
	return os;
}

#endif
