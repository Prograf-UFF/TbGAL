#include "hip/hip_runtime.h"
#ifndef OPERATIONS_CU
#define OPERATIONS_CU

#include "../../common.cu"

#include "../sparse_tensor/SparseTensor.cu"
#include <cusp/print.h>
#include <cusp/array1d.h>
#include <thrust/tuple.h>
#include "../sparse_tensor/utils.cu"
#include <vector>
#include <chrono>
#include <type_traits>
#include "../metric/metric.cu"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "multivector.cu"

#include <type_traits>
#include <set>

#include <boost/python.hpp>

namespace MultivectorOperations {

    Multivector GP_tensor(const Multivector &lhs, const Multivector &rhs, SparseTensor<IndexType, CoeffType, MemorySpace> *tensor);
    Multivector GP(const Multivector &lhs, const Multivector &rhs);
    Multivector ADD(const Multivector &lhs, const Multivector &rhs);
    Multivector SUB(const Multivector &lhs, const Multivector &rhs);
    Multivector SCP(const Multivector &lhs, const Multivector &rhs);
    Multivector SCP_tensor(const Multivector &lhs, const Multivector &rhs, SparseTensor<IndexType, CoeffType, MemorySpace> *tensor);
    Multivector OP(const Multivector &lhs, const Multivector &rhs);
    Multivector LCONT(const Multivector &lhs, const Multivector &rhs);
    Multivector RCONT(const Multivector &lhs, const Multivector &rhs);
    Multivector take_grade(const Multivector &m, IndexType grade);
    Multivector UNARY_MINUS(const Multivector &m);

    template<typename ScalarType, typename = typename std::enable_if<std::is_arithmetic<ScalarType>::value, ScalarType>::type>
    Multivector PROD (const Multivector &lhs, const ScalarType &rhs);
    template<typename ScalarType, typename = typename std::enable_if<std::is_arithmetic<ScalarType>::value, ScalarType>::type>
    Multivector ADD_SCALAR(const Multivector &lhs, const ScalarType &rhs);
    template<typename ScalarType, typename = typename std::enable_if<std::is_arithmetic<ScalarType>::value, ScalarType>::type>
    Multivector SUB_SCALAR(const Multivector &lhs, const ScalarType &rhs);
    template<typename ScalarType, typename = typename std::enable_if<std::is_arithmetic<ScalarType>::value, ScalarType>::type>
    Multivector R_SUB_SCALAR(const Multivector &lhs, const ScalarType &rhs);


    Multivector ADD(const Multivector &lhs, const Multivector &rhs) {
        return Multivector(lhs.getCore() + rhs.getCore());
    }

    template<typename ScalarType, typename>
    Multivector ADD_SCALAR(const Multivector &lhs, const ScalarType &rhs) {
        return ADD(lhs, Multivector(0, static_cast<CoeffType>(rhs)));
    }

    template<typename ScalarType, typename>
    Multivector PROD (const Multivector &lhs, const ScalarType &rhs) {
        return Multivector(static_cast<CoeffType>(rhs) * lhs.getCore());
    }

    Multivector UNARY_PLUS(const Multivector &m) {
        return Multivector(m.getCore());
    }

    Multivector UNARY_MINUS(const Multivector &m) {
        return PROD<CoeffType>(m, -1);
    }

    Multivector SUB(const Multivector &lhs, const Multivector &rhs) {
        return ADD(lhs, UNARY_MINUS(rhs));
    }

    template<typename ScalarType, typename>
    Multivector SUB_SCALAR(const Multivector &lhs, const ScalarType &rhs) {
        return ADD_SCALAR(lhs, -static_cast<CoeffType>(rhs));
    }

    template<typename ScalarType, typename>
    Multivector R_SUB_SCALAR(const Multivector &lhs, const ScalarType &rhs) {
        return ADD_SCALAR(UNARY_MINUS(lhs), static_cast<CoeffType>(rhs));
    }

    bool is_equals(const Multivector &lhs, const Multivector &rhs) {
        return (lhs.getCore() == rhs.getCore());
    }

    Multivector REVERSE(const Multivector &m) {
    	auto grade_2 = take_grade(m, 2);
    	auto new_mv = Multivector(m);
    	new_mv = ADD(new_mv, UNARY_MINUS(grade_2));
        new_mv = ADD(new_mv, UNARY_MINUS(grade_2));
        return new_mv;
    }

    Multivector INVOLUTION(const Multivector &m) {
    	auto grade_1 = take_grade(m, 1);
    	auto new_mv = Multivector(m);
    	new_mv = ADD(new_mv, UNARY_MINUS(grade_1));
        new_mv = ADD(new_mv, UNARY_MINUS(grade_1));
        return new_mv;
    }

    Multivector GP_tensor(const Multivector &lhs, const Multivector &rhs, SparseTensor<IndexType, CoeffType, MemorySpace> *tensor) {
      auto rhs_t = rhs.getCore().t();
    	return Multivector(SparseTensor<IndexType, CoeffType, MemorySpace>::tensor_dot(lhs.getCore(), rhs_t, *tensor).t());
    }

    Multivector GP(const Multivector &lhs, const Multivector &rhs) {
    	return GP_tensor(lhs, rhs, Multivector::get_GP_T());
    }

    Multivector SCP(const Multivector &lhs, const Multivector &rhs) {
    	Multivector ret = GP(lhs, rhs);
    	return take_grade(ret, static_cast<IndexType>(0));
    }

    Multivector SCP_tensor(const Multivector &lhs, const Multivector &rhs, SparseTensor<IndexType, CoeffType, MemorySpace> *tensor) {
    	Multivector ret = GP_tensor(lhs, rhs, tensor);
    	return take_grade(ret, 0);
    }

    Multivector OP(const Multivector &lhs, const Multivector &rhs) {
    	if (Multivector::get_OP_T() == NULL) {
    		OP_T = extract_tensor(Multivector::get_N(), Multivector::get_GP_T(), Operation::OUTER_PRODUCT); //TODO fix
    	}
    	auto r = GP_tensor(lhs, rhs, Multivector::get_OP_T());
    	return r;
    }

    Multivector LCONT(const Multivector &lhs, const Multivector &rhs) {
    	auto *T = extract_tensor(Multivector::get_N(), Multivector::get_GP_T(), Operation::LEFT_CONTRACTION); //TODO fix
    	auto r = GP_tensor(lhs, rhs, T);
    	return r;
    }


    Multivector RCONT(const Multivector &lhs, const Multivector &rhs) {
    	auto *T = extract_tensor(Multivector::get_N(), Multivector::get_GP_T(), Operation::RIGHT_CONTRACTION); //TODO fix
    	auto r = GP_tensor(lhs, rhs, T);
    	return r;
    }

    Multivector dot(const Multivector &lhs, const Multivector &rhs) {
    	auto *T = extract_tensor(Multivector::get_N(), Multivector::get_GP_T(), Operation::DOT_PRODUCT); //TODO fix
    	auto r = GP_tensor(lhs, rhs, T);
    	return r;
    }

    CoeffType SQR_NORM(const Multivector &m) {
    	auto r = GP(m, REVERSE(m));
    	return r.getCore().getCore().values[0];
    }

    Multivector INVERSE(const Multivector &m) {
    	return PROD<float>(REVERSE(m), (1.0/SQR_NORM(m)));
    }

    Multivector CONJUGATE(const Multivector &m) {
    	return REVERSE(INVOLUTION(m));
    }

    Multivector IGP(const Multivector &lhs, const Multivector &rhs) {
    	auto r = GP(lhs, INVERSE(rhs));
    	return r;
    }

    CoeffType NORM(const Multivector &m) {
    	return sqrt(SQR_NORM(m));
    }

    IndexType compute_max_arg_projection(const Multivector &m) {
    	auto all_idx = m.getComponentIndexes();
    	auto decompose_base = idx2ji<IndexType>(Multivector::get_N());
    	std::set<IndexType> all_basis;
    	for (IndexType idx : all_idx) {
    		thrust::tuple<IndexType, IndexType> pair = decompose_base(idx);
    		if (thrust::get<0>(pair) != 0) {
    			all_basis.insert(thrust::get<0>(pair));
    		}
    		all_basis.insert(thrust::get<1>(pair));
    	}
    	CoeffType max_norm = 0;
    	IndexType max_idx = 0;
    	for (IndexType i : all_basis) {
    		auto sqr_norm = SQR_NORM(LCONT(e(i), INVERSE(m)));
    		if (sqr_norm > max_norm) {
    			max_norm = sqr_norm;
    			max_idx = i;
    		}
    	}
    	return max_idx;
    }

    Multivector take_grade(const Multivector &m, IndexType grade) {
    	if (m.getCore().getRank() == 1) {
    	    cusp::array1d<IndexType, MemorySpace> indices = m.getCore().getCore().column_indices;
    	    cusp::array1d<CoeffType, MemorySpace> values = m.getCore().getCore().values;

    		cusp::array1d<IndexType, MemorySpace> new_indices(indices.size());
    		cusp::array1d<CoeffType, MemorySpace> new_values(values.size());

    		thrust::copy_if(indices.begin(), indices.end(), &indices[0], &new_indices[0], is_grade<IndexType>(Multivector::get_N(), grade));
    		thrust::copy_if(values.begin(), values.end(), &indices[0], &new_values[0], is_grade<IndexType>(Multivector::get_N(), grade));

    	    new_indices = m.getCore().erase_zeros<cusp::array1d<IndexType, MemorySpace>, cusp::array1d<CoeffType, MemorySpace>>(new_indices, new_values);
    	    new_values = m.getCore().erase_zeros<cusp::array1d<CoeffType, MemorySpace>, cusp::array1d<CoeffType, MemorySpace>>(new_values, new_values);

    		std::vector<IndexType> dense_shape = m.getCore().getDenseShape();

    		SparseTensor<IndexType, CoeffType, MemorySpace> new_core(new_indices, new_values, dense_shape, false);
    		return Multivector(new_core);
    	}
    	// TODO handler for exception
    	return NULL;
    }

	template<typename Container, typename std::enable_if<std::is_same<Container, std::vector<Multivector>>::value, Container>::type* = nullptr>
	Multivector getElementFromContainer(const Container &c, const IndexType &i) {
		return c[i];
	}

	template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type* = nullptr>
	Multivector getElementFromContainer(const Container &c, const IndexType &i) {
		return boost::python::extract<Multivector>(c[i]);
	}


	template<typename Container, typename std::enable_if<std::is_same<Container, std::vector<Multivector>>::value, Container>::type* = nullptr>
	void insertIntoContainer(Container &c, Multivector &m) {
		c.push_back(m);
	}

	template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type* = nullptr>
	void insertIntoContainer(Container &c, Multivector &m) {
		c.append(m);
	}

	template<typename Container>
    Container FACT_BLADE(Multivector &m) {
    	Container list;

    	// Perwass approach
    	// Multivector *A = new Multivector(*m);
    	// list.append(e(compute_max_arg_projection(A)));
    	// for (int i = 0; i < 1; i++) {
    	// 	auto *nj = LCONT(e(compute_max_arg_projection(A)), INVERSE(A));
    	// 	auto *nj_ = PROD(nj, 1.0 / NORM(nj));
    	// 	list.append(nj_);
    	// 	A = LCONT(INVERSE(nj_), A);
    	// }

    	//Fernandes et al approach
    	auto component_max_projection = m.get_component_max_projection();
    	// auto scalar = NORM(m);
    	auto temp = m;//PROD(m, 1.0 / scalar);

    	// list.append(scalar);
    	// list.append(temp);

    	auto all_idx = component_max_projection.getComponentIndexes();
    	auto decompose_base = idx2ji<IndexType>(Multivector::get_N());
    	std::set<IndexType> all_basis;
    	for (IndexType idx : all_idx) {
    		thrust::tuple<IndexType, IndexType> pair = decompose_base(idx);
    		if (thrust::get<0>(pair) != 0) {
    			all_basis.insert(thrust::get<0>(pair));
    		}
    		all_basis.insert(thrust::get<1>(pair));
    	}
    	std::vector<IndexType> basis(all_basis.begin(), all_basis.end());

    	for (IndexType i = 0; i < basis.size() - 1; i++) {
    		auto nj = LCONT(e(basis[i]), INVERSE(temp));
    		auto fatorj = PROD(nj, 1.0/NORM(nj));
			insertIntoContainer(list, fatorj);
			// list.append(fatorj);
    		temp = LCONT(INVERSE(fatorj), temp);
    	}
		insertIntoContainer(list, temp);
    	// list.append(temp);

        return list;
    }


	template<typename Container>
    Container FACT_VERSOR(const Multivector &V) {
    	Container list;

    	Multivector rev_V = REVERSE(V);
    	std::vector<int> grades_V = rev_V.get_grade();
    	// int k = 0;
    	while (!(grades_V.size() == 1 && grades_V[0] == 0)) {
    		// for (int i : grades_V) {
    			// std::cout << "GRADES_V: " << i << " IT: " << k << std::endl;
    		// }
    		// k++;
    		auto A = take_grade(rev_V, grades_V[grades_V.size() - 1]);
    		auto vectors = FACT_BLADE<Container>(A);
    		int i = 0;
    		while (dot(getElementFromContainer(vectors, i), getElementFromContainer(vectors, i)).getCore().getCore().values[0] == 0) {
    			i++;
    		}
    		Multivector n = getElementFromContainer(vectors, i);
    		// list.append(n);
			insertIntoContainer(list, n);
    		rev_V = GP(rev_V, n);
    		grades_V = rev_V.get_grade();
    	}
    	// list.append(rev_V);
		insertIntoContainer(list, rev_V);

    	return list;
    }
}

#endif
