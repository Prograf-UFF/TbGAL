#include "hip/hip_runtime.h"
#ifndef OPERATIONS_CU
#define OPERATIONS_CU

#include "../../common.cu"

#include "../sparse_tensor/SparseTensor.cu"
#include <cusp/print.h>
#include <cusp/array1d.h>
#include <thrust/tuple.h>
#include "../sparse_tensor/utils.cu"
#include <vector>
#include <chrono>
#include <type_traits>
#include "../metric/metric.cu"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "multivector.cu"

#include <type_traits>
#include <set>

#include <boost/python.hpp>
#include <boost/python/stl_iterator.hpp>

namespace MultivectorOperations {

    Multivector GP_tensor(const Multivector &lhs, const Multivector &rhs, SparseTensor<IndexType, CoeffType, MemorySpace> *tensor);
    // Multivector GP(const Multivector &lhs, const Multivector &rhs);

    template<typename Container, typename std::enable_if<std::is_same<Container, Multivector>::value, Container>::type* = nullptr>
    Container GP(const Container &lhs, const Container &rhs);

    template<typename Container, typename std::enable_if<std::is_same<Container, std::vector<Multivector>>::value, Container>::type* = nullptr>
  	Container GP(const Container &lhs, const Container &rhs);

  	template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type* = nullptr>
  	Container GP(const Container &lhs, const Container &rhs);

    template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type* = nullptr>
    Container process_factors(const Container &factors_A, const Multivector &factor_B);


    Multivector ADD(const Multivector &lhs, const Multivector &rhs);
    Multivector SUB(const Multivector &lhs, const Multivector &rhs);
    Multivector SCP(const Multivector &lhs, const Multivector &rhs);
    Multivector SCP_tensor(const Multivector &lhs, const Multivector &rhs, SparseTensor<IndexType, CoeffType, MemorySpace> *tensor);
    Multivector OP(const Multivector &lhs, const Multivector &rhs);
    Multivector LCONT(const Multivector &lhs, const Multivector &rhs);
    Multivector RCONT(const Multivector &lhs, const Multivector &rhs);
    Multivector take_grade(const Multivector &m, IndexType grade);
    Multivector UNARY_MINUS(const Multivector &m);

    template<typename ScalarType, typename = typename std::enable_if<std::is_arithmetic<ScalarType>::value, ScalarType>::type>
    Multivector PROD (const Multivector &lhs, const ScalarType &rhs);
    template<typename ScalarType, typename = typename std::enable_if<std::is_arithmetic<ScalarType>::value, ScalarType>::type>
    Multivector ADD_SCALAR(const Multivector &lhs, const ScalarType &rhs);
    template<typename ScalarType, typename = typename std::enable_if<std::is_arithmetic<ScalarType>::value, ScalarType>::type>
    Multivector SUB_SCALAR(const Multivector &lhs, const ScalarType &rhs);
    template<typename ScalarType, typename = typename std::enable_if<std::is_arithmetic<ScalarType>::value, ScalarType>::type>
    Multivector R_SUB_SCALAR(const Multivector &lhs, const ScalarType &rhs);


    template<typename Container, typename std::enable_if<std::is_same<Container, std::vector<Multivector>>::value, Container>::type* = nullptr>
  	Multivector getElementFromContainer(const Container &c, const IndexType &i);

  	template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type* = nullptr>
  	Multivector getElementFromContainer(const Container &c, const IndexType &i);


  	template<typename Container, typename std::enable_if<std::is_same<Container, std::vector<Multivector>>::value, Container>::type* = nullptr>
  	void insertIntoContainer(Container &c, Multivector &m);

  	template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type* = nullptr>
  	void insertIntoContainer(Container &c, Multivector &m);

    Multivector ADD(const Multivector &lhs, const Multivector &rhs) {
        return Multivector(lhs.getCore() + rhs.getCore());
    }

    template<typename ScalarType, typename>
    Multivector ADD_SCALAR(const Multivector &lhs, const ScalarType &rhs) {
        return ADD(lhs, Multivector(0, static_cast<CoeffType>(rhs)));
    }

    template<typename ScalarType, typename>
    Multivector PROD (const Multivector &lhs, const ScalarType &rhs) {
        return Multivector(static_cast<CoeffType>(rhs) * lhs.getCore());
    }

    Multivector UNARY_PLUS(const Multivector &m) {
        return Multivector(m.getCore());
    }

    Multivector UNARY_MINUS(const Multivector &m) {
        return PROD<CoeffType>(m, -1);
    }

    Multivector SUB(const Multivector &lhs, const Multivector &rhs) {
        return ADD(lhs, UNARY_MINUS(rhs));
    }

    template<typename ScalarType, typename>
    Multivector SUB_SCALAR(const Multivector &lhs, const ScalarType &rhs) {
        return ADD_SCALAR(lhs, -static_cast<CoeffType>(rhs));
    }

    template<typename ScalarType, typename>
    Multivector R_SUB_SCALAR(const Multivector &lhs, const ScalarType &rhs) {
        return ADD_SCALAR(UNARY_MINUS(lhs), static_cast<CoeffType>(rhs));
    }

    bool is_equals(const Multivector &lhs, const Multivector &rhs) {
        return (lhs.getCore() == rhs.getCore());
    }

    Multivector REVERSE(const Multivector &m) {
    	auto grade_2 = take_grade(m, 2);
    	auto new_mv = Multivector(m);
    	new_mv = ADD(new_mv, UNARY_MINUS(grade_2));
        new_mv = ADD(new_mv, UNARY_MINUS(grade_2));
        return new_mv;
    }

    Multivector INVOLUTION(const Multivector &m) {
    	auto grade_1 = take_grade(m, 1);
    	auto new_mv = Multivector(m);
    	new_mv = ADD(new_mv, UNARY_MINUS(grade_1));
        new_mv = ADD(new_mv, UNARY_MINUS(grade_1));
        return new_mv;
    }

    Multivector GP_tensor(const Multivector &lhs, const Multivector &rhs, SparseTensor<IndexType, CoeffType, MemorySpace> *tensor) {
      auto rhs_t = rhs.getCore().t();
    	return Multivector(SparseTensor<IndexType, CoeffType, MemorySpace>::tensor_dot(lhs.getCore(), rhs_t, *tensor).t());
    }

    Multivector GP(const Multivector &lhs, const Multivector &rhs) {
      return GP<Multivector>(lhs, rhs);
    }

    template<typename Container, typename std::enable_if<std::is_same<Container, Multivector>::value, Container>::type*>
    Container GP(const Container &lhs, const Container &rhs) {
      return GP_tensor(lhs, rhs, Multivector::get_GP_T());
    }

    template<typename Container, typename std::enable_if<std::is_same<Container, std::vector<Multivector>>::value, Container>::type*>
  	Container GP(const Container &lhs, const Container &rhs) {
  		return lhs;
  	}

    // Container new_factors_A;
    // Multivector inv_factor_B = INVERSE(factor_B);
    // Multivector new_factor = GP(getElementFromContainer(factors_A, 0), inv_factor_B);
    // if (new_factor.get_grade_blade() == 2 ) {
    //   new_factors_A += FACT_VERSOR<Container>(new_factor);
    // } else {
    //   new_factors_A.append(new_factor);
    // }
    // for (IndexType i = 1; i < boost::python::len(factors_A); i++) {
    //   new_factor = GP(GP(factor_B, getElementFromContainer(factors_A, i)), inv_factor_B);
    //   if (new_factor.get_grade_blade() ==2 ) {
    //     new_factors_A += FACT_VERSOR<Container>(new_factor);
    //   } else {
    //     new_factors_A.append(new_factor);
    //   }
    // }


  	template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type*>
  	Container GP(const Container &lhs, const Container &rhs) {
      Container new_factors = lhs;
      for (IndexType i = 0; i < boost::python::len(rhs); i++) {
        Container newest_factors;
        Container processed_factors = process_factors(new_factors, getElementFromContainer(rhs, i));
        for (IndexType k = 0; k < boost::python::len(processed_factors); k++) {
          newest_factors.append(getElementFromContainer(processed_factors, k));
        }
        new_factors = newest_factors;
      }
      return new_factors;
  	}

    Multivector SCP(const Multivector &lhs, const Multivector &rhs) {
    	Multivector ret = GP(lhs, rhs);
    	return take_grade(ret, static_cast<IndexType>(0));
    }

    Multivector SCP_tensor(const Multivector &lhs, const Multivector &rhs, SparseTensor<IndexType, CoeffType, MemorySpace> *tensor) {
    	Multivector ret = GP_tensor(lhs, rhs, tensor);
    	return take_grade(ret, 0);
    }

    Multivector OP(const Multivector &lhs, const Multivector &rhs) {
    	// if (Multivector::get_OP_T() == NULL) {
    		auto *T = extract_tensor(Multivector::get_N(), Multivector::get_GP_T(), Operation::OUTER_PRODUCT); //TODO fix
    	// }
    	auto r = GP_tensor(lhs, rhs, T);
    	return r;
    }

    Multivector LCONT(const Multivector &lhs, const Multivector &rhs) {
    	auto *T = extract_tensor(Multivector::get_N(), Multivector::get_GP_T(), Operation::LEFT_CONTRACTION); //TODO fix
    	auto r = GP_tensor(lhs, rhs, T);
    	return r;
    }


    Multivector RCONT(const Multivector &lhs, const Multivector &rhs) {
    	auto *T = extract_tensor(Multivector::get_N(), Multivector::get_GP_T(), Operation::RIGHT_CONTRACTION); //TODO fix
    	auto r = GP_tensor(lhs, rhs, T);
    	return r;
    }

    CoeffType native(const Multivector &m) {
      if (m.getCore().getCore().values.size() == 0) {
        return (CoeffType) 0;
      }
      return m.getCore().getCore().values[0];
    }

    CoeffType dot(const Multivector &lhs, const Multivector &rhs) {
    	auto *T = extract_tensor(Multivector::get_N(), Multivector::get_GP_T(), Operation::DOT_PRODUCT); //TODO fix
    	auto r = native(GP_tensor(lhs, rhs, T));
    	return r;
    }

    CoeffType SQR_NORM(const Multivector &m) {
    	auto r = GP(m, REVERSE(m));
    	return native(r);
    }

    Multivector INVERSE(const Multivector &m) {
    	return PROD<float>(REVERSE(m), (1.0/SQR_NORM(m)));
    }

    Multivector CONJUGATE(const Multivector &m) {
    	return REVERSE(INVOLUTION(m));
    }

    Multivector IGP(const Multivector &lhs, const Multivector &rhs) {
    	auto r = GP(lhs, INVERSE(rhs));
    	return r;
    }

    CoeffType NORM(const Multivector &m) {
    	return sqrt(SQR_NORM(m));
    }

    IndexType compute_max_arg_projection(const Multivector &m) {
    	auto all_idx = m.getComponentIndexes();
    	auto decompose_base = idx2ji<IndexType>(Multivector::get_N());
    	std::set<IndexType> all_basis;
    	for (IndexType idx : all_idx) {
    		thrust::tuple<IndexType, IndexType> pair = decompose_base(idx);
    		if (thrust::get<0>(pair) != 0) {
    			all_basis.insert(thrust::get<0>(pair));
    		}
    		all_basis.insert(thrust::get<1>(pair));
    	}
    	CoeffType max_norm = 0;
    	IndexType max_idx = 0;
    	for (IndexType i : all_basis) {
    		auto sqr_norm = SQR_NORM(LCONT(e(i), INVERSE(m)));
    		if (sqr_norm > max_norm) {
    			max_norm = sqr_norm;
    			max_idx = i;
    		}
    	}
    	return max_idx;
    }

    Multivector take_grade(const Multivector &m, IndexType grade) {
    	if (m.getCore().getRank() == 1) {
    	    cusp::array1d<IndexType, MemorySpace> indices = m.getCore().getCore().column_indices;
    	    cusp::array1d<CoeffType, MemorySpace> values = m.getCore().getCore().values;

    		cusp::array1d<IndexType, MemorySpace> new_indices(indices.size());
    		cusp::array1d<CoeffType, MemorySpace> new_values(values.size());

    		thrust::copy_if(indices.begin(), indices.end(), &indices[0], &new_indices[0], is_grade<IndexType>(Multivector::get_N(), grade));
    		thrust::copy_if(values.begin(), values.end(), &indices[0], &new_values[0], is_grade<IndexType>(Multivector::get_N(), grade));

    	    new_indices = m.getCore().erase_zeros<cusp::array1d<IndexType, MemorySpace>, cusp::array1d<CoeffType, MemorySpace>>(new_indices, new_values);
    	    new_values = m.getCore().erase_zeros<cusp::array1d<CoeffType, MemorySpace>, cusp::array1d<CoeffType, MemorySpace>>(new_values, new_values);

    		std::vector<IndexType> dense_shape = m.getCore().getDenseShape();

    		SparseTensor<IndexType, CoeffType, MemorySpace> new_core(new_indices, new_values, dense_shape, false);
    		return Multivector(new_core);
    	}
    	// TODO handler for exception
    	return NULL;
    }

	template<typename Container, typename std::enable_if<std::is_same<Container, std::vector<Multivector>>::value, Container>::type*>
	Multivector getElementFromContainer(const Container &c, const IndexType &i) {
		return c[i];
	}

	template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type*>
	Multivector getElementFromContainer(const Container &c, const IndexType &i) {
		return boost::python::extract<Multivector>(c[i]);
	}


	template<typename Container, typename std::enable_if<std::is_same<Container, std::vector<Multivector>>::value, Container>::type*>
	void insertIntoContainer(Container &c, Multivector &m) {
		c.push_back(m);
	}

	template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type*>
	void insertIntoContainer(Container &c, Multivector &m) {
		c.append(m);
	}

	template<typename Container>
    Container FACT_BLADE(Multivector &m) {
    	Container list;

    	// Perwass approach
    	// Multivector *A = new Multivector(*m);
    	// list.append(e(compute_max_arg_projection(A)));
    	// for (int i = 0; i < 1; i++) {
    	// 	auto *nj = LCONT(e(compute_max_arg_projection(A)), INVERSE(A));
    	// 	auto *nj_ = PROD(nj, 1.0 / NORM(nj));
    	// 	list.append(nj_);
    	// 	A = LCONT(INVERSE(nj_), A);
    	// }

    	//Fernandes et al approach
    	auto component_max_projection = m.get_component_max_projection();
    	auto scalar = NORM(m);
    	auto temp = PROD(m, 1.0 / scalar);

    	// list.append(scalar);
    	// list.append(temp);

    	auto all_idx = component_max_projection.getComponentIndexes();
    	auto decompose_base = idx2ji<IndexType>(Multivector::get_N());
    	std::set<IndexType> all_basis;
    	for (IndexType idx : all_idx) {
    		thrust::tuple<IndexType, IndexType> pair = decompose_base(idx);
    		if (thrust::get<0>(pair) != 0) {
    			all_basis.insert(thrust::get<0>(pair));
    		}
    		all_basis.insert(thrust::get<1>(pair));
    	}
    	std::vector<IndexType> basis(all_basis.begin(), all_basis.end());

    	for (IndexType i = 0; i < basis.size() - 1; i++) {
    		auto nj = LCONT(e(basis[i]), INVERSE(temp));
    		auto fatorj = PROD(nj, 1.0/NORM(nj));
			insertIntoContainer(list, fatorj);
			// list.append(fatorj);
    		temp = LCONT(INVERSE(fatorj), temp);
    	}
		insertIntoContainer(list, temp);
    if (abs(scalar - 1) > epsilon) {
      auto norm = Multivector(0, scalar);
      insertIntoContainer(list, norm);
    }
    	// list.append(temp);

        return list;
    }


	template<typename Container>
    Container FACT_VERSOR(const Multivector &V) {
    	Container list;

    	Multivector rev_V = REVERSE(V);
    	std::vector<int> grades_V = rev_V.get_grade();
    	// int k = 0;
    	while (!(grades_V.size() == 1 && grades_V[0] == 0)) {
    		// for (int i : grades_V) {
    			// std::cout << "GRADES_V: " << i << " IT: " << k << std::endl;
    		// }
    		// k++;
    		auto A = take_grade(rev_V, grades_V[grades_V.size() - 1]);
    		auto vectors = FACT_BLADE<Container>(A);
    		int i = 0;
    		while (dot(getElementFromContainer(vectors, i), getElementFromContainer(vectors, i)) == 0) {
    			i++;
    		}
    		Multivector n = getElementFromContainer(vectors, i);
    		// list.append(n);
			insertIntoContainer(list, n);
    		rev_V = GP(rev_V, n);
    		grades_V = rev_V.get_grade();
    	}
    	// list.append(rev_V);
      if (abs(NORM(rev_V)) - 1 > epsilon) {
        insertIntoContainer(list, rev_V);
      }

    	return list;
    }


    template< typename T >
    std::vector< T > to_std_vector( const boost::python::list &iterable ) {
        return std::vector< T >( boost::python::stl_input_iterator< T >( iterable ),
                                 boost::python::stl_input_iterator< T >( ) );
    }


    template<typename Container, typename std::enable_if<std::is_same<Container, boost::python::list>::value, Container>::type*>
    Container process_factors(const Container &factors_A, const Multivector &factor_B) {
      std::vector<Multivector> std_factors_A = to_std_vector<Multivector>(factors_A);
      if ((std_factors_A[std_factors_A.size()-1]^factor_B) == (e(1)^e(1))) {
        if (std_factors_A[std_factors_A.size()-1] == factor_B) {
          std_factors_A.pop_back();
          return boost::python::list(std_factors_A);
        }
        std_factors_A[std_factors_A.size()-1] = GP(std_factors_A[std_factors_A.size()-1], factor_B);
        return boost::python::list(std_factors_A);
      }

      std::vector<Multivector> new_factors_A;
      Multivector inv_factor_B = INVERSE(factor_B);
      Multivector new_factor = GP(std_factors_A[0], inv_factor_B);
      if (new_factor.get_grade_blade() == 2 ) {
        auto factors_from_new_factor = FACT_VERSOR<std::vector<Multivector>>(new_factor);
        new_factors_A.insert(std::end(new_factors_A), std::begin(factors_from_new_factor), std::end(factors_from_new_factor));
      } else {
        new_factors_A.push_back(new_factor);
      }
      for (IndexType i = 1; i < std_factors_A.size(); i++) {
        new_factor = GP(GP(factor_B, std_factors_A[i]), inv_factor_B);
        if (new_factor.get_grade_blade() ==2 ) {
          auto factors_from_new_factor = FACT_VERSOR<std::vector<Multivector>>(new_factor);
          new_factors_A.insert(std::end(new_factors_A), std::begin(factors_from_new_factor), std::end(factors_from_new_factor));
        } else {
          new_factors_A.push_back(new_factor);
        }
      }
      new_factors_A[new_factors_A.size() - 1] = PROD(static_cast<Multivector>(new_factors_A[new_factors_A.size() - 1]), NORM(factor_B));
      return boost::python::list(new_factors_A);
    }

}

#endif
