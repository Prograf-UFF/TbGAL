#include <boost/python.hpp>


class teste {
public:
    static void set_var(int var) {
        var_ = var;
    }
    int v2;
    teste() {
        if (var_ == 0) {
            throw 0;
        }
        this->v2 = var_ * 2;
    }
    static int var_;
};

int teste::var_ = 0;


/*
static void set_var(int var) {
    teste::var = var;
}
*/
BOOST_PYTHON_MODULE(teste)
{

    namespace python = boost::python;

    python::class_<teste, boost::noncopyable>("Teste")
        .def("set_var", &teste::set_var)
            .staticmethod("set_var");
//    python::def("set_var", set_var<int>);


}
