#include "hip/hip_runtime.h"
#include <multivector.cu>
#include <operations.cu>
#include <metric.cu>
#include <math.h>
#include "benchmark_utils.cu"
#include <algorithm>
#include <iterator>
#include <chrono>
#include <fstream>
#include <iostream>

using namespace std;
using namespace MultivectorOperations;

#define PI acos(-1)

void run_loops(std::vector<Multivector> &R, std::vector<Multivector> &S, const int &RUNS, double &avg_time, double &std_dev) {
	std::vector<double> times(RUNS);
	for (int i = 0; i < RUNS; i++) {
		auto start = std::chrono::high_resolution_clock::now();
		GP(R, S);
		auto finish = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> elapsed_seconds = finish - start;
		times[i] = elapsed_seconds.count();
		avg_time += times[i];
	}
	avg_time /= RUNS;
	for (int i = 0; i < RUNS; i++) {
		std_dev += (times[i] - avg_time)*(times[i] - avg_time);
	}
	std_dev /= (RUNS - 1);
	std_dev = sqrt(std_dev);
}

void run_iterations(const int &DIMS, const int &RUNS) {
	std::vector<Multivector> R;
	std::vector<Multivector> S;
	auto results = ofstream("results.csv", std::ios::app);
	for (IndexType r = 3; r <= DIMS; r++) {
		process_lines(read_lines("R", DIMS, r), R);
		for (IndexType s = 3; s <= DIMS; s++) {
			process_lines(read_lines("S", DIMS, s), S);

			double avg_time = 0;
			double std_dev = 0;

			run_loops(R, S, RUNS, avg_time, std_dev);
			S.clear();
			S.resize(0);
			S.shrink_to_fit();
			results << std::to_string(DIMS) << "," << std::to_string(r) << "," << std::to_string(s) << "," << std::to_string(avg_time) << "," << std::to_string(std_dev) << std::endl;
		}
		R.clear();
		R.resize(0);
		R.shrink_to_fit();
	}
	results.close();
}

int main() {

	IndexType MAX_DIMS = 100;
	int RUNS = 10;
	ofstream results ("results.csv");
	results << "N,R,S,avg_time,std_dev_time" << std::endl;
	results.close();
	for (IndexType DIMS = 3; DIMS <= MAX_DIMS; DIMS++) {
		Multivector::set_N(DIMS);
		generate_T<EuclideanMetric>(EuclideanMetric());
		run_iterations(DIMS, RUNS);
		delete_T();
	}


	// auto M = (e(1)^e(2)) + (e(1)^e(3));
	// auto m = MultivectorOperations::FACT_BLADE<std::vector<Multivector>>(M);

	// std::vector<string> lines = read_lines("R", 150, 6);


	// for (auto &i : k) {
	// 	std::cout << i << std::endl;
	// }
	// std::copy(v.begin(), v.end(), ostream_iterator<Multivector>(std::cout, "\n"));


	// auto output = MultivectorOperations::GP(getElementFromContainer(m, 0), getElementFromContainer(m, 1));
	//
	// std::cout << "output" << output << std::endl;
	// std::cout << "M" << M << std::endl;
	// std::cout << "Is output equals to M ? " << (output == M) << std::endl;
	//
	// auto B = (e(1)^e(2))+(e(1)^e(3)+(e(2)^e(3)));
	// B = B * (1.0 / MultivectorOperations::NORM(B));
	//
	// auto V = cos(PI/4) - ((+sin(PI/4.0)*B));
	// auto v = MultivectorOperations::FACT_VERSOR<std::vector<Multivector>>(V);
	// output = MultivectorOperations::GP(MultivectorOperations::GP(getElementFromContainer(v, 0), getElementFromContainer(v, 1)), getElementFromContainer(v, 2));
	// std::cout << "output" << output << std::endl;
	// std::cout << "M" << V << std::endl;
	// std::cout << "Is output equals to V ? " << (output == V) << std::endl;



    return 0;
}
