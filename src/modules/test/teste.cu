#include "hip/hip_runtime.h"
#include <multivector.cu>
#include <operations.cu>
#include <metric.cu>
#include <math.h>

using namespace std;
using namespace MultivectorOperations;

#define PI acos(-1)

int main() {

	Multivector::set_N(5);
	auto metric = EuclideanMetric();
	generate_T<EuclideanMetric>(metric);

	auto M = (e(1)^e(2)) + (e(1)^e(3));
	auto m = MultivectorOperations::FACT_BLADE<std::vector<Multivector>>(M);


	auto output = MultivectorOperations::GP(getElementFromContainer(m, 0), getElementFromContainer(m, 1));

	std::cout << "output" << output << std::endl;
	std::cout << "M" << M << std::endl;
	std::cout << "Is output equals to M ? " << (output == M) << std::endl;

	auto B = (e(1)^e(2))+(e(1)^e(3)+(e(2)^e(3)));
	B = B * (1.0 / MultivectorOperations::NORM(B));

	auto V = cos(PI/4) - ((+sin(PI/4.0)*B));
	auto v = MultivectorOperations::FACT_VERSOR<std::vector<Multivector>>(V);
	output = MultivectorOperations::GP(MultivectorOperations::GP(getElementFromContainer(v, 0), getElementFromContainer(v, 1)), getElementFromContainer(v, 2));
	std::cout << "output" << output << std::endl;
	std::cout << "M" << V << std::endl;
	std::cout << "Is output equals to V ? " << (output == V) << std::endl;



    return 0;
}
